#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"

#define IN_TILE_DIM 32
#define OUT_TILE_DIM ((IN_TILE_DIM) - 2*(FILTER_RADIUS))

__constant__ float filter_c[FILTER_DIM][FILTER_DIM];

__global__ void convolution_tiled_kernel(float* input, float* output, unsigned int width, unsigned int height) {

     __shared__ float cov[IN_TILE_DIM][IN_TILE_DIM];
     unsigned int row = blockIdx.y*blockDim.y + threadIdx.y;
     unsigned int col = blockIdx.x*blockDim.x + threadIdx.x;
     float sum = 0.0f;
     for(unsigned int tile = 0; tile < (width + TILE_DIM -1 ) / TILE_DIM; ++tile) {
        if((row >= 0) && (row< height) && (col >= 0) && (col < width) ) {
        cov[threadIdx.y][threadIdx.x]= cov[row*width + tile*TILE_DIM + threadIdx.x];
    }
    else{
     cov[threadIdx.y][threadIdx.x]=0;
     }
     if(threadIdx.y < TILE_DIM && threadIdx.x < TILE_DIM){
        for(i = 0; i < FILTER_DIM; i++) {
            for(j = 0; j < FILTER_DIM; j++) { 
                output += filter_c_[i][j] * cov[i+threadIdx.y][j+threadIdx.x];
} }
     
    output[outRow*width + outCol] = sum;
     }
     
     









}

void copyFilterToGPU(float filter[][FILTER_DIM]) {

    // Copy filter to constant memory

    hipMemcpyToSymbol(HIP_SYMBOL(filter_c), filter, FILTER_DIM*FILTER_DIM*sizeof(float));

}

void convolution_tiled_gpu(float* input_d, float* output_d, unsigned int width, unsigned int height) {

    // Call kernel

    dim3 numThreadsPerBlock(OUT_TILE_DIM, OUT_TILE_DIM);
    dim3 numBlocks((width + OUT_TILE_DIM - 1)/OUT_TILE_DIM, (height + OUT_TILE_DIM - 1)/OUT_TILE_DIM);
    convolution_tiled_kernel <<< numBlocks, numThreadsPerBlock >>> (input_d, output_d, width, height);



}

